#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int device;
    hipDeviceProp_t prop;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    std::cout << "max shared memory (per block): "
              << prop.sharedMemPerBlock / 1000 << " KB" << std::endl;
    std::cout << "max shared memory (per SM): "
              << prop.sharedMemPerMultiprocessor / 1000 << " KB" << std::endl;

    return 0;
}

